#include "hip/hip_runtime.h"
#include "floydWarshallZeroCopy.h"
#include <thrust/device_vector.h>
#include <chrono>

#include <hip/hip_runtime.h>

#define BLOCKSIZE 256

// Doesn't do negative edges!!
__global__
void iterKernel(int k, int *distances, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y;

    // If we're over the edge of the matrix return
    if ((col >= N) || (distances[N * row + k] == INT_MAX) || (distances[k * N + col] == INT_MAX)) {
        return;
    }

    // Otherwise, calculate the distance
    int candidateBetterDistance = distances[N * row + k] + distances[k * N + col];
    if (candidateBetterDistance < distances[N * row + col])
        distances[N * row + col] = candidateBetterDistance;
}

int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void floydWarshallZeroCopy(thrust::host_vector<int>& h_vec, double* copyTimings, double* execTimings)
{
        int* hostData = thrust::raw_pointer_cast(h_vec.data());
        int N = sqrt(h_vec.size());
        int size = N * N * sizeof(int);

        // Track subtask time
        auto timeInit = std::chrono::high_resolution_clock::now();

        hipHostAlloc((void**)&hostData, size, hipHostMallocMapped);
        int* cudaData;
        hipHostGetDevicePointer((void**)&cudaData, (void*)hostData, 0);

        // Device memory allocated
        auto timeHtD = std::chrono::high_resolution_clock::now();

        for (int k = 0; k < N; k++) {
            iterKernel << < dim3(iDivUp(N, BLOCKSIZE), N), BLOCKSIZE >> > (k, cudaData, N);
        }

        // Calculations complete
        auto timeExec = std::chrono::high_resolution_clock::now();

        std::chrono::duration<double, std::milli> hostToDevice = timeHtD - timeInit;
        std::cout << "Copying data from host to device took " << hostToDevice.count() << " ms." << std::endl;

        *copyTimings += hostToDevice.count();

        std::chrono::duration<double, std::milli> exec = timeExec - timeHtD;
        std::cout << "Executing calculations took " << exec.count() << " ms." << std::endl;

        *execTimings += exec.count();

        hipHostFree(hostData);

 

    
    
}