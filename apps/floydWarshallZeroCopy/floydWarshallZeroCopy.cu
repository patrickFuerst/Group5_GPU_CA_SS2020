#include "hip/hip_runtime.h"
#include "floydWarshallZeroCopy.h"
#include <thrust/device_vector.h>
#include <chrono>

#include <hip/hip_runtime.h>

#define BLOCKSIZE 256

// Doesn't do negative edges!!
__global__
void iterKernel(int k, int *distances, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y;

    // If we're over the edge of the matrix return
    if ((col >= N) || (distances[N * row + k] == INT_MAX / 2) || (distances[k * N + col] == INT_MAX / 2)) {
        return;
    }

    // Otherwise, calculate the distance
    int candidateBetterDistance = distances[N * row + k] + distances[k * N + col];
    if (candidateBetterDistance < distances[N * row + col])
        distances[N * row + col] = candidateBetterDistance;
}

int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void floydWarshallZeroCopy(thrust::host_vector<int>& h_vec)
{
    int * hostData = thrust::raw_pointer_cast(h_vec.data());
    int N = sqrt(h_vec.size());

    // Track subtask time
    auto timeInit = std::chrono::high_resolution_clock::now();
    
    int * cudaData;
    hipMalloc(&cudaData, N * N * sizeof(int));
    hipMemcpy(cudaData, hostData, N * N * sizeof(int), hipMemcpyHostToDevice);
    
    // Device memory allocated
    auto timeHtD = std::chrono::high_resolution_clock::now();
    
    for (int k = 0; k < N; k++) {
        iterKernel<<< dim3(iDivUp(N, BLOCKSIZE), N), BLOCKSIZE >>> (k, cudaData, N);
    }

    // Calculations complete
	auto timeExec = std::chrono::high_resolution_clock::now();

    // Get results back
    hipMemcpy(hostData, cudaData, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Results moved to host
    auto timeDtH = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> hostToDevice = timeHtD - timeInit;
    std::cout << "Copying data from host to device took " << hostToDevice.count() << " ms." << std::endl;

    std::chrono::duration<double, std::milli> exec = timeExec - timeHtD;
    std::cout << "Executing calculations took " << exec.count() << " ms." << std::endl;

    std::chrono::duration<double, std::milli> deviceToHost = timeDtH - timeExec;
    std::cout << "Copying results from device to host took " << deviceToHost.count() << " ms." << std::endl;


    std::cout << "Hello!" << std::endl;
}