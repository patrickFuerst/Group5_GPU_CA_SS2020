#include "hip/hip_runtime.h"

#include "floydWarshallCuda.h"
#include <thrust/device_vector.h>


#define BLOCKSIZE 256

// Doesn't do negative edges!!
__global__
void cu_FloydWarshall(int k, int *distances, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y;

    // If we're over the edge of the matrix return
    if (col >= N) return;

    int arrayIndex = N * row + col;

    // Each block loads the entire BLOCK column into shared memory
    __shared__ int current;
    // This is done by thread #0
    if(threadIdx.x == 0)
    	current = distances[N * row + k];
    // The rest of the threads should wait
    __syncthreads();
    
    // If the current distance is INF, return
    if (current == INT_MAX / 2)
    return;

    // If the follow up distance is INF, return
    int next = distances[k * N + col];
    if(next == INT_MAX / 2)
    return;

    int candidateBetterDistance = current + next;
    if (candidateBetterDistance < distances[arrayIndex])
        distances[arrayIndex] = candidateBetterDistance;
}



int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void floydWarshallCuda(thrust::host_vector<int>& h_vec)
{
    // Transfer graph to GPU
    //int matrixSize = graph.mNumVertices * graph.mNumVertices;
    //int* cudaDistances;
    //hipMalloc(&cudaDistances, matrixSize * sizeof(int));
    //hipMemcpy(cudaDistances, m.data, matrixSize * sizeof(int), hipMemcpyHostToDevice);
    int N = sqrt(h_vec.size());

    thrust::device_vector<int> d_vec = h_vec;
    thrust::device_ptr< int > d_ptr = d_vec.data();
    // For each node, iterate distances
    for (int k = 0; k < N; k++) {
        cu_FloydWarshall<<< dim3(iDivUp(N, BLOCKSIZE), N), BLOCKSIZE >>> ( k, thrust::raw_pointer_cast(d_ptr), N );
    }

    // Get results back
    h_vec = d_vec;


}